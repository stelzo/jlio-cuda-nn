#include <memory.cuh>
#include <point.cuh>

namespace jlio
{
    void malloc(void **ptr, size_t size)
    {
        CHECK_CUDA_ERROR(hipMallocManaged(ptr, size));
    }

    void memset(void *ptr, int value, size_t count)
    {
        CHECK_CUDA_ERROR(hipMemset(ptr, value, count));
    }

    void memcpy(void *dst, const void *src, size_t count, int kind)
    {
        ::hipMemcpyKind _kind;
        switch (kind)
        {
        case 0:
            _kind = ::hipMemcpyHostToHost;
            break;
        case 1:
            _kind = ::hipMemcpyHostToDevice;
            break;
        case 2:
            _kind = ::hipMemcpyDeviceToHost;
            break;
        case 3:
            _kind = ::hipMemcpyDeviceToDevice;
            break;
        case 4:
            _kind = ::hipMemcpyDefault;
            break;
        default:
            _kind = ::hipMemcpyDefault;
        }
        CHECK_CUDA_ERROR(hipMemcpy(dst, src, count, _kind));
        // dst = src; // TODO only on jetson: no need to copy to and from device when using unified memory; but just assigning the pointer would make free fail
    }

    void free(void *ptr)
    {
        if (ptr == NULL)
        {
            return;
        }

        CHECK_CUDA_ERROR(hipFree(ptr));
    }

} // namespace jlio
